/*
 * Copyright 2009 Colin Percival, 2011 ArtForz, 2011-2013 pooler
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 * This file was originally written by Colin Percival as part of the Tarsnap
 * online backup system.
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#ifndef __linux__
#include <machine/endian.h>
#else
#include <endian.h>
#endif

__host__ __device__ inline uint32_t swab32(uint32_t x) {
  return (((x & 0x000000ffUL) << 24) |
	  ((x & 0x0000ff00UL) << 8) |
	  ((x & 0x00ff0000UL) >> 8) |
	  ((x & 0xff000000UL) >> 24));
}

__host__ void sha256_init(uint32_t *state);
__device__ void dev_sha256_init(uint32_t *state);
void sha256_transform(uint32_t *state, const uint32_t *block, int swap);
__device__ void dev_sha256_transform(uint32_t *state, const uint32_t *block, int swap);

static const uint32_t keypad[12] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x00000280
};
static const uint32_t innerpad[11] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x000004a0
};
static const uint32_t outerpad[8] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0x00000300
};
static const uint32_t finalblk[16] = {
	0x00000001, 0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x00000620
};

__constant__ static const uint32_t dev_keypad[12] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x00000280
};
__constant__ static const uint32_t dev_innerpad[11] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x000004a0
};
__constant__ static const uint32_t dev_outerpad[8] = {
	0x80000000, 0, 0, 0, 0, 0, 0, 0x00000300
};
__constant__ static const uint32_t dev_finalblk[16] = {
	0x00000001, 0x80000000, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0x00000620
};

/*
 * write_8_as_uint4 and read_8_as_uint4 are helper functions to reduce memory
 * transactions by reading in larger word sizes.
 */

__device__ inline void write_8_as_uint4(const uint32_t *s, uint32_t *d32) {
  uint4 *d = (uint4 *)d32;
    uint4 t;
    t.x = s[0]; t.y = s[1]; t.z = s[2]; t.w = s[3];
    d[0] = t;
    t.x = s[4]; t.y = s[5]; t.z = s[6]; t.w = s[7];
    d[1] = t;
}

__device__ inline void read_8_as_uint4(const uint32_t *s32, uint32_t *d) {
  uint4 *s = (uint4 *)s32;
  uint4 t;
  t = s[0];
  d[0] = t.x; d[1] = t.y; d[2] = t.z; d[3] = t.w;
  t = s[1];
  d[4] = t.x; d[5] = t.y; d[6] = t.z; d[7] = t.w;
}



static inline void HMAC_SHA256_80_init(const uint32_t *key,
	uint32_t *tstate, uint32_t *ostate)
{
	uint32_t ihash[8];
	uint32_t pad[16];
	int i;

	/* tstate is assumed to contain the midstate of key */
	memcpy(pad, key + 16, 16);
	memcpy(pad + 4, keypad, 48);
	sha256_transform(tstate, pad, 0);
	memcpy(ihash, tstate, 32);

	sha256_init(ostate);
	for (i = 0; i < 8; i++)
		pad[i] = ihash[i] ^ 0x5c5c5c5c;
	for (; i < 16; i++)
		pad[i] = 0x5c5c5c5c;
	sha256_transform(ostate, pad, 0);

	sha256_init(tstate);
	for (i = 0; i < 8; i++)
		pad[i] = ihash[i] ^ 0x36363636;
	for (; i < 16; i++)
		pad[i] = 0x36363636;
	sha256_transform(tstate, pad, 0);
}

__device__ static inline void dev_HMAC_SHA256_80_init(const uint32_t *key,
	uint32_t *tstate, uint32_t *ostate)
{
	uint32_t ihash[8];
	uint32_t pad[16];
	int i;

	/* tstate is assumed to contain the midstate of key */
	pad[0] = key[16]; pad[1] = key[17]; pad[2] = key[18]; pad[3] = key[19];
	for (i = 0; i < 12; i++) {
	  pad[4+i] = dev_keypad[i];
	}
	dev_sha256_transform(tstate, pad, 0);
	for (i = 0; i < 8; i++) {
	  ihash[i] = tstate[i];
	}

	dev_sha256_init(ostate);
	for (i = 0; i < 8; i++)
		pad[i] = ihash[i] ^ 0x5c5c5c5c;
	for (; i < 16; i++)
		pad[i] = 0x5c5c5c5c;
	dev_sha256_transform(ostate, pad, 0);

	dev_sha256_init(tstate);
	for (i = 0; i < 8; i++)
		pad[i] = ihash[i] ^ 0x36363636;
	for (; i < 16; i++)
		pad[i] = 0x36363636;
	dev_sha256_transform(tstate, pad, 0);
}

static inline void PBKDF2_SHA256_80_128(const uint32_t *tstate,
	const uint32_t *ostate, const uint32_t *salt, uint32_t *output)
{
	uint32_t istate[8], ostate2[8];
	uint32_t ibuf[16], obuf[16];
	int i, j;

	memcpy(istate, tstate, 32);
	sha256_transform(istate, salt, 0);
	
	memcpy(ibuf, salt + 16, 16);
	memcpy(ibuf + 5, innerpad, 44);
	memcpy(obuf + 8, outerpad, 32);

	for (i = 0; i < 4; i++) {
		memcpy(obuf, istate, 32);
		ibuf[4] = i + 1;
		sha256_transform(obuf, ibuf, 0);

		memcpy(ostate2, ostate, 32);
		sha256_transform(ostate2, obuf, 0);
		for (j = 0; j < 8; j++)
			output[8 * i + j] = swab32(ostate2[j]);
	}
}

__device__ static inline void dev_PBKDF2_SHA256_80_128(const uint32_t *tstate,
	const uint32_t *ostate, const uint32_t *salt, uint32_t *output)
{
	uint32_t istate[8], ostate2[8];
	uint32_t ibuf[16], obuf[16];
	int i, j;

	memcpy(istate, tstate, 32);
	dev_sha256_transform(istate, salt, 0);
	
	memcpy(ibuf, salt + 16, 16);
	memcpy(ibuf + 5, dev_innerpad, 44);
	memcpy(obuf + 8, dev_outerpad, 32);

	for (i = 0; i < 4; i++) {
		memcpy(obuf, istate, 32);
		ibuf[4] = i + 1;
		dev_sha256_transform(obuf, ibuf, 0);

		memcpy(ostate2, ostate, 32);
		dev_sha256_transform(ostate2, obuf, 0);
		for (j = 0; j < 8; j++)
			output[8 * i + j] = swab32(ostate2[j]);
	}
}


static inline void PBKDF2_SHA256_128_32(uint32_t *tstate, uint32_t *ostate,
	const uint32_t *salt, uint32_t *output)
{
	uint32_t buf[16];
	int i;
	
	sha256_transform(tstate, salt, 1);
	sha256_transform(tstate, salt + 16, 1);
	sha256_transform(tstate, finalblk, 0);
	memcpy(buf, tstate, 32);
	memcpy(buf + 8, outerpad, 32);

	sha256_transform(ostate, buf, 0);
	for (i = 0; i < 8; i++)
		output[i] = swab32(ostate[i]);
}

__device__ static inline void dev_PBKDF2_SHA256_128_32(uint32_t *tstate, uint32_t *ostate,
	const uint32_t *salt)
{
	uint32_t buf[16];
	
	dev_sha256_transform(tstate, salt, 1);
	dev_sha256_transform(tstate, salt + 16, 1);
	dev_sha256_transform(tstate, dev_finalblk, 0);
	memcpy(buf, tstate, 32);
	memcpy(buf + 8, dev_outerpad, 32);

	dev_sha256_transform(ostate, buf, 0);
// output moved into CleanupHashN to use write_8 function
}


/*********** sha2.c ********/

/*
 * Copyright 2011 ArtForz
 * Copyright 2011-2013 pooler
 *
 * This program is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License as published by the Free
 * Software Foundation; either version 2 of the License, or (at your option)
 * any later version.  See COPYING for more details.
 */

#include <string.h>
#include <inttypes.h>

#if defined(__arm__) && defined(__APCS_32__)
#define EXTERN_SHA256
#endif

__constant__ const uint32_t dev_sha256_h[8] = {
	0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
	0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
};

static const uint32_t sha256_h[8] = {
	0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a,
	0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19
};


__constant__ static const uint32_t sha256_k[64] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
	0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
	0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
	0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
	0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
	0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
	0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
	0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
	0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};


static const uint32_t host_sha256_k[64] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
	0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
	0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
	0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
	0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
	0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
	0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
	0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
	0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
};

__device__ inline void dev_sha256_init(uint32_t *state)
{
  for (int i = 0; i < 8; i++) {
    state[i] = dev_sha256_h[i];
  }
}

__host__ void sha256_init(uint32_t *state)
{
  for (int i = 0; i < 8; i++) {
    state[i] = sha256_h[i];
  }
}

/* Elementary functions used by SHA256 */
#define Ch(x, y, z)     ((x & (y ^ z)) ^ z)
#define Maj(x, y, z)    ((x & (y | z)) | (y & z))
#define ROTR(x, n)      ((x >> n) | (x << (32 - n)))
#define S0(x)           (ROTR(x, 2) ^ ROTR(x, 13) ^ ROTR(x, 22))
#define S1(x)           (ROTR(x, 6) ^ ROTR(x, 11) ^ ROTR(x, 25))
#define s0(x)           (ROTR(x, 7) ^ ROTR(x, 18) ^ (x >> 3))
#define s1(x)           (ROTR(x, 17) ^ ROTR(x, 19) ^ (x >> 10))

/* SHA256 round function */
#define RND(a, b, c, d, e, f, g, h, k) \
	do { \
		t0 = h + S1(e) + Ch(e, f, g) + k; \
		t1 = S0(a) + Maj(a, b, c); \
		d += t0; \
		h  = t0 + t1; \
	} while (0)

/* Adjusted round function for rotating state */
#define RNDr(S, W, i) \
	RND(S[(64 - i) % 8], S[(65 - i) % 8], \
	    S[(66 - i) % 8], S[(67 - i) % 8], \
	    S[(68 - i) % 8], S[(69 - i) % 8], \
	    S[(70 - i) % 8], S[(71 - i) % 8], \
	    W[i] + sha256_k[i])

#define hRNDr(S, W, i) \
	RND(S[(64 - i) % 8], S[(65 - i) % 8], \
	    S[(66 - i) % 8], S[(67 - i) % 8], \
	    S[(68 - i) % 8], S[(69 - i) % 8], \
	    S[(70 - i) % 8], S[(71 - i) % 8], \
	    W[i] + host_sha256_k[i])

#ifndef EXTERN_SHA256


/*
 * SHA256 block compression function.  The 256-bit state is transformed via
 * the 512-bit input block to produce a new state.
 */
void sha256_transform(uint32_t *state, const uint32_t *block, int swap)
{
	uint32_t W[64];
	uint32_t S[8];
	uint32_t t0, t1;
	int i;

	/* 1. Prepare message schedule W. */
	if (swap) {
		for (i = 0; i < 16; i++)
			W[i] = swab32(block[i]);
	} else
		memcpy(W, block, 64);
	for (i = 16; i < 64; i += 2) {
		W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15];
	}

	/* 2. Initialize working variables. */
	memcpy(S, state, 32);

	/* 3. Mix. */
	hRNDr(S, W,  0);
	hRNDr(S, W,  1);
	hRNDr(S, W,  2);
	hRNDr(S, W,  3);
	hRNDr(S, W,  4);
	hRNDr(S, W,  5);
	hRNDr(S, W,  6);
	hRNDr(S, W,  7);
	hRNDr(S, W,  8);
	hRNDr(S, W,  9);
	hRNDr(S, W, 10);
	hRNDr(S, W, 11);
	hRNDr(S, W, 12);
	hRNDr(S, W, 13);
	hRNDr(S, W, 14);
	hRNDr(S, W, 15);
	hRNDr(S, W, 16);
	hRNDr(S, W, 17);
	hRNDr(S, W, 18);
	hRNDr(S, W, 19);
	hRNDr(S, W, 20);
	hRNDr(S, W, 21);
	hRNDr(S, W, 22);
	hRNDr(S, W, 23);
	hRNDr(S, W, 24);
	hRNDr(S, W, 25);
	hRNDr(S, W, 26);
	hRNDr(S, W, 27);
	hRNDr(S, W, 28);
	hRNDr(S, W, 29);
	hRNDr(S, W, 30);
	hRNDr(S, W, 31);
	hRNDr(S, W, 32);
	hRNDr(S, W, 33);
	hRNDr(S, W, 34);
	hRNDr(S, W, 35);
	hRNDr(S, W, 36);
	hRNDr(S, W, 37);
	hRNDr(S, W, 38);
	hRNDr(S, W, 39);
	hRNDr(S, W, 40);
	hRNDr(S, W, 41);
	hRNDr(S, W, 42);
	hRNDr(S, W, 43);
	hRNDr(S, W, 44);
	hRNDr(S, W, 45);
	hRNDr(S, W, 46);
	hRNDr(S, W, 47);
	hRNDr(S, W, 48);
	hRNDr(S, W, 49);
	hRNDr(S, W, 50);
	hRNDr(S, W, 51);
	hRNDr(S, W, 52);
	hRNDr(S, W, 53);
	hRNDr(S, W, 54);
	hRNDr(S, W, 55);
	hRNDr(S, W, 56);
	hRNDr(S, W, 57);
	hRNDr(S, W, 58);
	hRNDr(S, W, 59);
	hRNDr(S, W, 60);
	hRNDr(S, W, 61);
	hRNDr(S, W, 62);
	hRNDr(S, W, 63);

	/* 4. Mix local working variables into global state */
	for (i = 0; i < 8; i++)
		state[i] += S[i];
}

__device__ void dev_sha256_transform(uint32_t *state, const uint32_t *block, int swap)
{
	uint32_t W[64];
	uint32_t S[8];
	uint32_t t0, t1;
	int i;

	/* 1. Prepare message schedule W. */
	if (swap) {
	  for (i = 0; i < 16; i++)
	    W[i] = swab32(block[i]);
	} else {
	  for (i = 0; i < 16; i++) {
	    W[i] = block[i];
	  }
	}

	for (i = 16; i < 64; i += 2) {
	  W[i]   = s1(W[i - 2]) + W[i - 7] + s0(W[i - 15]) + W[i - 16];
		W[i+1] = s1(W[i - 1]) + W[i - 6] + s0(W[i - 14]) + W[i - 15];
	}

	/* 2. Initialize working variables. */
	  for (i = 0; i < 8; i++) { S[i] = state[i]; }

	/* 3. Mix. */
	RNDr(S, W,  0);
	RNDr(S, W,  1);
	RNDr(S, W,  2);
	RNDr(S, W,  3);
	RNDr(S, W,  4);
	RNDr(S, W,  5);
	RNDr(S, W,  6);
	RNDr(S, W,  7);
	RNDr(S, W,  8);
	RNDr(S, W,  9);
	RNDr(S, W, 10);
	RNDr(S, W, 11);
	RNDr(S, W, 12);
	RNDr(S, W, 13);
	RNDr(S, W, 14);
	RNDr(S, W, 15);
	RNDr(S, W, 16);
	RNDr(S, W, 17);
	RNDr(S, W, 18);
	RNDr(S, W, 19);
	RNDr(S, W, 20);
	RNDr(S, W, 21);
	RNDr(S, W, 22);
	RNDr(S, W, 23);
	RNDr(S, W, 24);
	RNDr(S, W, 25);
	RNDr(S, W, 26);
	RNDr(S, W, 27);
	RNDr(S, W, 28);
	RNDr(S, W, 29);
	RNDr(S, W, 30);
	RNDr(S, W, 31);
	RNDr(S, W, 32);
	RNDr(S, W, 33);
	RNDr(S, W, 34);
	RNDr(S, W, 35);
	RNDr(S, W, 36);
	RNDr(S, W, 37);
	RNDr(S, W, 38);
	RNDr(S, W, 39);
	RNDr(S, W, 40);
	RNDr(S, W, 41);
	RNDr(S, W, 42);
	RNDr(S, W, 43);
	RNDr(S, W, 44);
	RNDr(S, W, 45);
	RNDr(S, W, 46);
	RNDr(S, W, 47);
	RNDr(S, W, 48);
	RNDr(S, W, 49);
	RNDr(S, W, 50);
	RNDr(S, W, 51);
	RNDr(S, W, 52);
	RNDr(S, W, 53);
	RNDr(S, W, 54);
	RNDr(S, W, 55);
	RNDr(S, W, 56);
	RNDr(S, W, 57);
	RNDr(S, W, 58);
	RNDr(S, W, 59);
	RNDr(S, W, 60);
	RNDr(S, W, 61);
	RNDr(S, W, 62);
	RNDr(S, W, 63);

	/* 4. Mix local working variables into global state */
	for (i = 0; i < 8; i++)
		state[i] += S[i];
}


#endif /* EXTERN_SHA256 */

